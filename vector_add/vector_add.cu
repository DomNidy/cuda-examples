
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000 * 1000 // num of blocks

__global__ void blockAdd(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void threadAdd(int *a, int *b, int *c)
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void printVec(const void *vec, size_t elementSize)
{
    printf("[ ");
    for (int i = 0; i < N; i++)
    {
        printf("%d ", *((int *)vec + i));
    }
    printf("]\n");
}

int main(void)
{
    int size = sizeof(int);

    // host copies of a,b,c
    // allocate memory to store vectors
    int *a = (int *)calloc(N, size);
    int *b = (int *)calloc(N, size);
    int *c = (int *)calloc(N, size);

    int *d_a, *d_b, *d_c; // device copies of a,b,c

    if (a == NULL || b == NULL || c == NULL)
    {
        printf("Memory allocation failed\n");
        return 1;
    }

    // assign some random values to our vectors
    for (int i = 0; i < N; i++)
    {
        *(a + i) = (5 + i) % 3;
        *(b + i) = (5 + i) % 7;
    }

    // printf("Vectors initialized to:\n");
    // printVec(a, size);
    // printVec(b, size);
    // printVec(c, size);

    // allocate space on the device for our vector copies
    // but remember, we cant dereference these from host code or pass to host code
    // we can only use these pointers in our kernels running on blocks on the gpu
    hipMalloc((void **)&d_a, N * size);
    hipMalloc((void **)&d_b, N * size);
    hipMalloc((void **)&d_c, N * size);

    // copy inputs to the device
    // this moves data into the region of memory we allocated
    // on the gpu earlier (with cudaMalloc)
    hipMemcpy(d_a, a, N * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * size, hipMemcpyHostToDevice);

    // launch kernel with N blocks
    blockAdd<<<N, 1>>>(d_a, d_b, d_c);

    // launch kernel with 1 block split into N threads
    // threadAdd<<<1, N>>>(d_a, d_b, d_c);

    // copy back to host, reading from the region of memory `d_c`
    // we allocated on the device, into the region on the host `c`
    hipMemcpy(c, d_c, N * size, hipMemcpyDeviceToHost);

    // free gpu memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // printf("Result from threadAdd kernel:\n");
    // printVec(c, size);
    int sum = 0;
    for (int i = 0; i < N; i++)
    {
        sum += c[i];
    }
    printf("cuda=%d\n", sum);

    return 0;
}